#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <time.h>

#define ARRAY_SIZE 128
#define THREADS_PER_BLOCK 512

// Kernel for bitonic sort
__global__ void bitonic_sort(int* dev_array, int step, int stage) {
    // Thread index
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    // Length of the sub-sequence
    int sub_sequence_length = pow(2, step - stage + 1);
    // Skip for comparisons
    int jump = pow(2, step - stage);

    // Check if thread should participate in comparisons
    if (tid % sub_sequence_length < jump) {
        // If tid/2^step is even, sort ascendingly
        if ((tid / (int)pow(2, step)) % 2 == 0) {
            if (dev_array[tid] > dev_array[tid + jump]) {
                // Swap elements if needed
                int temp = dev_array[tid];
                dev_array[tid] = dev_array[tid + jump];
                dev_array[tid + jump] = temp;
            }
        }
        // If tid/2^step is odd, sort descendingly
        else {
            if (dev_array[tid] < dev_array[tid + jump]) {
                // Swap elements if needed
                int temp = dev_array[tid];
                dev_array[tid] = dev_array[tid + jump];
                dev_array[tid + jump] = temp;
            }
        }
    }
}

// Print an array with a title in a formatted manner
void printArray(const char* title, int* arr, int size) {
    // Print the title and the opening bracket
    printf("%s: [", title);

    // Iterate through the array elements
    for (int i = 0; i < size; i++) {
        // Add a comma before each element (except the first one)
        if (i != 0) {
            printf(", ");
        }

        // Print the array element
        printf("%d", arr[i]);
    }

    // Print the closing bracket and a newline character
    printf("]\n");
}

int main(void) {
    // Check if array size is a power of 2
    if (!(log2(ARRAY_SIZE) == (int)log2(ARRAY_SIZE))) {
        printf("Array size must be a power of 2 for Bitonic Sort!");
        return 0;
    }

    int host_array[ARRAY_SIZE];

    // Assign random numbers to each element in the array
    for (int i = 0; i < ARRAY_SIZE; i++) {
        host_array[i] = rand() % 300;
    }

    // Print unsorted array
    printArray("Random unsorted array", host_array, ARRAY_SIZE);


    // Define device array
    int* dev_array;
    int size = ARRAY_SIZE * sizeof(int);
    // Allocate memory on the device
    hipMalloc(&dev_array, size);
    // Copy data from host to device
    hipMemcpy(dev_array, host_array, size, hipMemcpyHostToDevice);

    // Bitonic sort algorithm Steps
    for (int step = 1; step <= log2(ARRAY_SIZE); step += 1) {
        // Iterate through Stages
        for (int stage = 1; stage <= step; stage += 1) {
            // Call the bitonic_sort kernel
            bitonic_sort << <((ARRAY_SIZE + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK), THREADS_PER_BLOCK >> > (dev_array, step, stage);
        }
    }

    // Copy sorted array back to host
    hipMemcpy(host_array, dev_array, size, hipMemcpyDeviceToHost);
    // Free device memory
    hipFree(dev_array);

    // Print sorted array
    printArray("\nSorted array", host_array, ARRAY_SIZE);


    return 0;
}
